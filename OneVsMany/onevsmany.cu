#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "choice_methods.h"
#include "read_write.h"

using namespace std;

void StartTimer(StopWatchInterface *timer)
{
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
}
void StopTimer(StopWatchInterface *timer)
{
	sdkStopTimer(&timer);
	printf("[GPU] processing time : %f (ms)\n", sdkGetTimerValue(&timer));
}

void CalculateUserIndexes(int userCount, int* hostUserRows, int* &deviceUserRows, int* &hostBasicIndexes, int* &deviceBasicIndexes, int* &hostUserIndexes, int* &deviceUserIndexes, float* &hostDistancesUser, float* &deviceDistancesUser, int &totalSelfUsersMemory)
{
	int userIndexesMemory = userCount * sizeof(int);
	hostBasicIndexes = (int*)malloc(userIndexesMemory);
	hostUserIndexes = (int*)malloc(userIndexesMemory);
	int lastIndex = 0;
	int lastBasicIndex = 0;
	for (int i = 0; i < userCount; i++)
	{
		hostUserIndexes[i] = lastIndex;
		hostBasicIndexes[i] = lastBasicIndex;
		int rows = hostUserRows[i];
		int memory = rows * rows;
		lastIndex += memory;
		lastBasicIndex += rows;
	}
	totalSelfUsersMemory = lastIndex * sizeof(float);
	hostDistancesUser = (float*)malloc(totalSelfUsersMemory);

	checkCudaErrors(hipMalloc((void **)&deviceUserRows, userIndexesMemory));
	checkCudaErrors(hipMemcpy(deviceUserRows, hostUserRows, userIndexesMemory, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **)&deviceUserIndexes, userIndexesMemory));
	checkCudaErrors(hipMemcpy(deviceUserIndexes, hostUserIndexes, userIndexesMemory, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **)&deviceBasicIndexes, userIndexesMemory));
	checkCudaErrors(hipMemcpy(deviceBasicIndexes, hostBasicIndexes, userIndexesMemory, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **)&deviceDistancesUser, totalSelfUsersMemory));
	if (!hostDistancesUser || !deviceDistancesUser)
		printf("Memory allocation for user failed\n");
}

__host__ __device__ inline static
float Distance(float* dataTab, int columnCount, int x, int y)
{
	int column = 1;
	float xVal = dataTab[x * columnCount + column];
	float yVal = dataTab[y * columnCount + column];
	return abs(xVal - yVal);
}

__global__ void
DistancesPerUser(int* userRows, int* resultUserIndexes, int* basicUserIndexes, int columnCount, float* dataTab, float* distancesUser)
{
	int threadId = threadIdx.x;
	int blockId = blockIdx.x;
	int blockSize = blockDim.x;
	int rows = userRows[blockId];
	int cellsToCalculate = rows * rows;
	int resultStartIndex = resultUserIndexes[blockId];
	int basicStartIndex = basicUserIndexes[blockId];
	//printf("Block=%d, Tid=%d, Rows=%d, StartIndex=%d\n", blockId, threadId, rows, startIndex);
	for (int i = threadId; i < cellsToCalculate; i += blockSize)
	{
		int x = basicStartIndex + i % rows;
		int y = basicStartIndex + i / rows;
		distancesUser[resultStartIndex + i] = Distance(dataTab, columnCount, x, y);
	}
}

__global__ void
oneVsManyRowWise(float* dataTab, int rowCount, int columnCount, float* resultTab, int* usersRowsIndexes)
{
	const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= rowCount) return;
	int myIndex = usersRowsIndexes[tid];
	int resultShift = rowCount * tid;

	for (int i = 0; i < rowCount; i++)
	{
		int otherIndex = usersRowsIndexes[i];
		resultTab[resultShift + i] = Distance(dataTab, columnCount, myIndex, otherIndex);
	}
}

__global__ void
oneVsManyUserBlock(float* dataTab, int rowCount, int columnCount, float* resultTab, int* usersRowsIndexes, int* userStartIndexes, int userCount)
{
	int threadId = threadIdx.x;
	int blockId = blockIdx.x;
	int blockSize = blockDim.x;
	int userRows;
	if (blockId == userCount - 1)
		userRows = rowCount - userStartIndexes[userCount - 1];
	else 
		userRows = userStartIndexes[blockId + 1] - userStartIndexes[blockId];

	int cellsToCalculate = userRows * rowCount;
	int userStartIndex = userStartIndexes[blockId];
	int resultShift = userStartIndex * rowCount;
	//printf("Block=%d, Tid=%d, Rows=%d, Shift=%d\n", blockId, threadId, userRows, resultShift);
	for (int i = threadId; i < cellsToCalculate; i += blockSize)
	{
		int x = usersRowsIndexes[(resultShift + i) % rowCount];
		int y = usersRowsIndexes[(resultShift + i) / rowCount];
		resultTab[resultShift + i] = Distance(dataTab, columnCount, x, y);
	}
}


int main(int argc, char **argv)
{
	// Config
	auto config = ReadConfig();
	int THREADS_PER_BLOCK = stoi(config["THREADS_PER_BLOCK"]);
	int ROW_SELECTION_METHOD = stoi(config["ROW_SELECTION_METHOD"]);
	int ONE_VS_MANY_METHOD = stoi(config["ONE_VS_MANY_METHOD"]);
	LoadChoiceMethodsConfigValues();

	// Main
    srand(7312);
	int tpb = THREADS_PER_BLOCK;
    StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);

	float* hostDataTab;
    float * deviceDataTab;
	int* hostUserTab;
	int* deviceUserTab;	
	int rowCount;
	int columnCount;

	int* hostUserRows;
	int* deviceUserRows;
	int userCount;
	float* hostDistancesUser;
	float* deviceDistancesUser;
	int* hostUserResultIndexes;
	int* deviceUserResultIndexes;
	int* hostUserBasicIndexes;
	int* deviceUserBasicIndexes;
	int totalSelfUsersMemory;
	vector<string> ids;
	int* sessionNumbers;

	ReadData(rowCount, columnCount, userCount, hostDataTab, hostUserTab, hostUserRows, ids, sessionNumbers);

	// Memory allocation
	int dataMemory = sizeof(float) * rowCount * columnCount;
	int userMemory = sizeof(int) * rowCount;
    checkCudaErrors(hipMalloc((void **)& deviceDataTab, dataMemory));
    checkCudaErrors(hipMemcpy(deviceDataTab, hostDataTab, dataMemory, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **)& deviceUserTab, userMemory));
	checkCudaErrors(hipMemcpy(deviceUserTab, hostUserTab, userMemory, hipMemcpyHostToDevice));
	
	if(!hostDataTab || !deviceDataTab || !hostUserTab || !deviceUserTab) printf("Memory allocation failed\n");

	//Distances Per User
	CalculateUserIndexes(userCount, hostUserRows, deviceUserRows, hostUserBasicIndexes, deviceUserBasicIndexes, hostUserResultIndexes, deviceUserResultIndexes, hostDistancesUser, deviceDistancesUser, totalSelfUsersMemory);
	//PrintTab(hostUserRows, userCount);
	//PrintTab(hostUserResultIndexes, userCount);

	//StartTimer(timer);
	DistancesPerUser << < userCount, tpb >> > (deviceUserRows, deviceUserResultIndexes, deviceUserBasicIndexes, columnCount, deviceDataTab, deviceDistancesUser);
	hipDeviceSynchronize();
	//StopTimer(timer);
	checkCudaErrors(hipMemcpy(hostDistancesUser, deviceDistancesUser, totalSelfUsersMemory, hipMemcpyDeviceToHost));
	//PrintUsersSelfDistances(userCount, hostUserRows, hostUserResultIndexes, hostDistancesUser);

	//You need to specify below values for usage in oneVsMany
	int sumOfSelectedRows;
	int * usersRowsIndexes;
	int * userStartIndexes = (int*)malloc(userCount * sizeof(int));
	
	//Selecting rows from groups
	switch (ROW_SELECTION_METHOD)
	{
		case 1:
			sumOfSelectedRows = rowCount;
			usersRowsIndexes = ChooseAll(rowCount);
			userStartIndexes = hostUserBasicIndexes;
			break;
		case 2:
			sumOfSelectedRows = ChooseRandom(usersRowsIndexes, userStartIndexes, hostUserRows, hostUserBasicIndexes, userCount);
			break;
		case 3:
			sumOfSelectedRows = ChooseHeuristic(usersRowsIndexes, userStartIndexes, userCount, hostUserRows, hostUserResultIndexes, hostDistancesUser, hostUserBasicIndexes);
			break;
		case 4:
			sumOfSelectedRows = ChooseCharacteristicAreas(usersRowsIndexes, userStartIndexes, userCount, hostUserRows, hostUserResultIndexes, hostDistancesUser, hostUserBasicIndexes);
			break;
		case 5:
			sumOfSelectedRows = ChooseWithoutDistant(usersRowsIndexes, userStartIndexes, userCount, hostUserRows, hostUserResultIndexes, hostDistancesUser, hostUserBasicIndexes);
			break;
	}
	//PrintSelectedRows(sumOfSelectedRows, userCount, usersRowsIndexes, userStartIndexes);
	
	//Memory Allocation for OneVsMany
	int resultMemory = sizeof(float) * sumOfSelectedRows * sumOfSelectedRows;
	float * hostResultTab = (float*)malloc(resultMemory);
	float * deviceResultTab;
	checkCudaErrors(hipMalloc((void **)&deviceResultTab, resultMemory));
	int * deviceUsersRowsIndexes;
	int usersRowsIndexesMemory = sumOfSelectedRows * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&deviceUsersRowsIndexes, usersRowsIndexesMemory));
	checkCudaErrors(hipMemcpy(deviceUsersRowsIndexes, usersRowsIndexes, usersRowsIndexesMemory, hipMemcpyHostToDevice));

	int * deviceUserStartIndexes;
	int userStartIndexesMemory = userCount * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&deviceUserStartIndexes, userStartIndexesMemory));
	checkCudaErrors(hipMemcpy(deviceUserStartIndexes, userStartIndexes, userStartIndexesMemory, hipMemcpyHostToDevice));
	if (!hostResultTab || !deviceResultTab || !deviceUsersRowsIndexes || !deviceUserStartIndexes) printf("Memory allocation failed\n");

	// OneVsMany Calculations
	int numberOfBlocks;
    StartTimer(timer);
	switch (ONE_VS_MANY_METHOD) {
	case 1:
		numberOfBlocks = sumOfSelectedRows / tpb + 1;
		oneVsManyRowWise << < numberOfBlocks, tpb >> > (deviceDataTab, sumOfSelectedRows, columnCount, deviceResultTab, deviceUsersRowsIndexes);
		break;
	case 2:
		numberOfBlocks = userCount;
		oneVsManyUserBlock << < numberOfBlocks, tpb >> > (deviceDataTab, sumOfSelectedRows, columnCount, deviceResultTab, deviceUsersRowsIndexes, deviceUserStartIndexes, userCount);
		break;
	}
	hipDeviceSynchronize();
	StopTimer(timer);
	checkCudaErrors(hipMemcpy(hostResultTab, deviceResultTab, resultMemory, hipMemcpyDeviceToHost));

	//Print Results
	//PrintTab2D(sumOfSelectedRows, hostResultTab);
	WriteResults(hostResultTab, sumOfSelectedRows, userCount, usersRowsIndexes, userStartIndexes, ids, sessionNumbers);

	//Free memory and timer
	sdkDeleteTimer(&timer);
	free(hostDataTab); free(hostUserTab); free(usersRowsIndexes); free(hostDistancesUser);free(hostUserBasicIndexes); free(hostUserResultIndexes);
	hipFree(deviceDataTab); hipFree(deviceUserTab); hipFree(deviceUsersRowsIndexes); hipFree(deviceDistancesUser);hipFree(deviceUserBasicIndexes); hipFree(deviceUserResultIndexes);
    getLastCudaError("Kernel execution failed");
}
