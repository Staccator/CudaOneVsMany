#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <random>
#include <string>
#include <iterator>
#include <algorithm>
#include <vector>
#include <numeric>
#include <functional>

#include "choice_methods.h"
#include "read_write.h"

int MAX_ROWS_PER_USER;
// Method 4 & 5
int BUCKET_COUNT;
float BUCKET_SIZE;
// Method 4
int CHARACTERISTIC_BUCKETS_COUNT;
// Method 5
float BUCKET_SIZE_TO_BE_REJECTED;

using namespace std;

void LoadChoiceMethodsConfigValues()
{
	auto config = ReadConfig();

	MAX_ROWS_PER_USER = stoi(config["MAX_ROWS_PER_USER"]);
	BUCKET_COUNT = stoi(config["BUCKET_COUNT"]);
	BUCKET_SIZE = stof(config["BUCKET_SIZE"]);
	CHARACTERISTIC_BUCKETS_COUNT = stoi(config["CHARACTERISTIC_BUCKETS_COUNT"]);
	BUCKET_SIZE_TO_BE_REJECTED = stof(config["BUCKET_SIZE_TO_BE_REJECTED"]);
}

void PrintSelectedRows(int checkedRowsCount, int userCount, int* usersRowsIndexes, int* userStartIndexes)
{
	cout << "Selected Rows" << endl;
	for (int i = 0; i < userCount; i++)
	{
		cout << userStartIndexes[i] << " ";
	} cout << endl;

	int j = 0;
	for (int i = 0; i < checkedRowsCount; i++)
	{
		if (i == userStartIndexes[j])
		{
			cout << endl;
			j++;
		}
		cout << usersRowsIndexes[i] << " ";
	} cout << endl;
}

int* ChooseAll(int rowCount)
{
	int* result = (int*)malloc(rowCount * sizeof(int));
	for (int i = 0; i < rowCount; i++)
	{
		result[i] = i;
	}
	return result;
}

int ChooseRandom(int* &usersRowsIndexes, int* &userStartIndexes, int* hostUserRows, int* hostUserBasicIndexes, int userCount)
{
	int usersRowsIndexesMemory = 0;
	int rowsPerUser = MAX_ROWS_PER_USER;
	int lastIndex = 0;
	for (int i = 0; i < userCount; i++)
	{
		int selectedRowsCount = min(rowsPerUser, hostUserRows[i]);
		usersRowsIndexesMemory += selectedRowsCount;
		userStartIndexes[i] = lastIndex;
		lastIndex += selectedRowsCount;
	}

	usersRowsIndexes = (int*)malloc(usersRowsIndexesMemory * sizeof(int));
	for (int i = 0; i < userCount; i++)
	{
		int selectedRowsCount = min(rowsPerUser, hostUserRows[i]);

		std::vector<int> v;
		for (int j = 0; j < hostUserRows[i]; j++)
		{
			v.push_back(hostUserBasicIndexes[i] + j);
		}
		std::random_device rd;
		std::mt19937 g(rd());
		std::shuffle(v.begin(), v.end(), g);
		std::copy(v.begin(), v.begin() + selectedRowsCount, usersRowsIndexes + userStartIndexes[i]);
	}

	return usersRowsIndexesMemory;
}

int ChooseHeuristic(int* &usersRowsIndexes, int* &userStartIndexes, int userCount, int* userRows, int* startIndexes, float* distancesUser, int* hostUserBasicIndexes)
{
	int usersRowsIndexesMemory = 0;
	int rowsPerUser = MAX_ROWS_PER_USER;
	int lastIndex = 0;
	for (int i = 0; i < userCount; i++)
	{
		int selectedRowsCount = min(rowsPerUser, userRows[i]);
		usersRowsIndexesMemory += selectedRowsCount;
		userStartIndexes[i] = lastIndex;
		lastIndex += selectedRowsCount;
	}

	usersRowsIndexes = (int*)malloc(usersRowsIndexesMemory * sizeof(int));
	for (int i = 0; i < userCount; i++)
	{
		int rows = userRows[i];
		int startIndex = startIndexes[i];
		std::vector<float> sort_values;
		std::vector<int> v;
		for (int x = 0; x < rows; x++)
		{
			float sum = 0;
			for (int y = 0; y < rows; y++)
			{
				sum += distancesUser[startIndex + x * rows + y];
			}
			sort_values.push_back(sum);
			v.push_back(hostUserBasicIndexes[i] + x);
		}

		std::vector<std::size_t> index(v.size());
		std::iota(index.begin(), index.end(), 0);
		std::sort(index.begin(), index.end(), [&](size_t a, size_t b) { return sort_values[a] < sort_values[b]; });
		int selectedRowsCount = min(rowsPerUser, userRows[i]);
		for (int z = 0; z < selectedRowsCount; z++)
		{
			usersRowsIndexes[userStartIndexes[i] + z] = v[index[z]];
		}
	}

	return usersRowsIndexesMemory;
}

int ChooseWithoutDistant(int* &usersRowsIndexes, int* &userStartIndexes, int userCount, int* userRows, int* startIndexes, float* distancesUser, int* hostUserBasicIndexes)
{
	int bucketCount = BUCKET_COUNT;
	float bucketSize = BUCKET_SIZE;
	int* buckets = (int*)malloc(bucketCount * sizeof(int));

	int lastIndex = 0;
	vector<int> resultIndexes;

	for (int i = 0; i < userCount; i++)
	{
		int rows = userRows[i];
		int startIndex = startIndexes[i];
		for (int j = 0; j < bucketCount; j++)
		{
			buckets[j] = 0;
		}

		for (int x = 0; x < rows; x++)
		{
			for (int y = 0; y < rows; y++)
			{
				if (x == y) continue;
				float distance = distancesUser[startIndex + x * rows + y];
				int index = distance / bucketSize;
				if (index < bucketCount) 
					{ buckets[index] += 1; }
			}
		}

		int selectedRowsCount = 0;
		for (int x = 0; x < rows; x++)
		{
			bool isCorrect = true;
			for (int y = 0; y < rows; y++)
			{
				if (x == y) continue;
				float distance = distancesUser[startIndex + x * rows + y];
				int index = distance / bucketSize;
				if (index < bucketCount)
				{
					if (buckets[index] <= BUCKET_SIZE_TO_BE_REJECTED) {
						isCorrect = false; break;
					}
				}
				else
				{
					isCorrect = false; break;
				}
			}
			if (isCorrect) 
			{
				if (selectedRowsCount >= MAX_ROWS_PER_USER) continue;

				resultIndexes.push_back(hostUserBasicIndexes[i] + x);
				selectedRowsCount += 1;
			}
		}

		userStartIndexes[i] = lastIndex;
		lastIndex += selectedRowsCount;
	}

	usersRowsIndexes = (int*)malloc(resultIndexes.size() * sizeof(int));
	copy(resultIndexes.begin(), resultIndexes.end(), usersRowsIndexes);
	return resultIndexes.size();
}

int ChooseCharacteristicAreas(int* &usersRowsIndexes, int* &userStartIndexes, int userCount, int* userRows, int* startIndexes, float* distancesUser, int* hostUserBasicIndexes)
{
	int bucketCount = BUCKET_COUNT;
	float bucketSize = BUCKET_SIZE;
	int* buckets = (int*)malloc(bucketCount * sizeof(int));

	int lastIndex = 0;
	vector<int> resultIndexes;

	for (int i = 0; i < userCount; i++)
	{
		int rows = userRows[i];
		int startIndex = startIndexes[i];
		for (int j = 0; j < bucketCount; j++) buckets[j] = 0;

		for (int x = 0; x < rows; x++)
		{
			for (int y = 0; y < rows; y++)
			{
				if (x == y) continue;
				float distance = distancesUser[startIndex + x * rows + y];
				int index = distance / bucketSize;
				if (index < bucketCount)
				{
					buckets[index] += 1;
				}
			}
		}

		int selectedRowsCount = 0;
		std::vector<std::size_t> index(bucketCount);
		std::iota(index.begin(), index.end(), 0);
		std::sort(index.begin(), index.end(), [&](size_t a, size_t b) { return buckets[a] > buckets[b]; });
		for (int j = 0; j < bucketCount; j++) buckets[j] = 0;
		for (int j = 0; j < CHARACTERISTIC_BUCKETS_COUNT; j++) {
			buckets[index[j]] = 1;
		}
		//for (int j = 0; j < bucketCount; j++) cout << buckets[j] << " "; cout << endl;

		for (int x = 0; x < rows; x++)
		{
			bool isCorrect = false;
			for (int y = 0; y < rows; y++)
			{
				if (x == y) continue;
				float distance = distancesUser[startIndex + x * rows + y];
				int index = distance / bucketSize;
				if (index < bucketCount)
				{
					if (buckets[index] == 1)
					{
						isCorrect = true; break;
					}
				}
			}
			if (isCorrect)
			{
				if (selectedRowsCount >= MAX_ROWS_PER_USER) continue;

				resultIndexes.push_back(hostUserBasicIndexes[i] + x);
				selectedRowsCount += 1;
			}
		}

		userStartIndexes[i] = lastIndex;
		lastIndex += selectedRowsCount;
	}

	//for (int i = 0; i < resultIndexes.size(); i++) cout << resultIndexes[i] << " "; cout << endl;
	usersRowsIndexes = (int*)malloc(resultIndexes.size() * sizeof(int));
	copy(resultIndexes.begin(), resultIndexes.end(), usersRowsIndexes);
	return resultIndexes.size();
}